#include "DeviceFunctions.cuh"
#include <cub/device/device_radix_sort.cuh>
#include <cub/device/device_scan.cuh> 



namespace CUBLIB
{
	void Sort(uint*& keys, uint*& objectsIDs, uint size)
	{
		/// Allocate the neccesary 
		uint* sortedKeys;
		hipMalloc(&sortedKeys, sizeof(uint) * size);
		uint *sortedIDs;
		hipMalloc(&sortedIDs, sizeof(uint) * size);

		// Allocate temporary storage for sorting
		size_t  temp_storage_bytes = 0;
		void    *d_temp_storage = NULL;
		// Find the temp storage
		hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, keys, sortedKeys, objectsIDs, sortedIDs, size);
		// Allocate temporary storage
		hipMalloc(&d_temp_storage, temp_storage_bytes);
		// Start the sort
		hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, keys, sortedKeys, objectsIDs, sortedIDs, size);


		hipFree(keys);
		hipFree(objectsIDs);

		objectsIDs = sortedIDs;
		keys = sortedKeys;
	}


	uint* PrefixSum(uint* keys, uint size)
	{
		uint* keys_out;
		hipMalloc(&keys_out, sizeof(uint) * size);
		// Allocate temporary storage
		size_t  temp_storage_bytes = 0;
		void    *d_temp_storage = NULL;
		//// Find the temp storage
		hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, keys, keys_out, size);
		// Allocate temporary storage for inclusive prefix sum
		hipMalloc(&d_temp_storage, temp_storage_bytes);
		// Run inclusive prefix sum
		hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, keys, keys_out, size);

		return keys_out;
	}
}

